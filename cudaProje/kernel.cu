#include "hip/hip_runtime.h"
 #include <cstdio>
 #include <cstdlib>
#include <cctype>
#include <hip/hip_runtime.h>
#include ""
 #include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

__global__ void
vectorAdd(const float *A, const float *B, float *C, int numElements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < numElements)
	{
		C[i] = A[i] + B[i];
	}
}


int
main(void)
{

	hipError_t err = hipSuccess;

	int numElements = 50000;
	size_t size = numElements * sizeof(float);
	printf("[%d elemanl� vektor ekleme]\n", numElements);

	float *h_A = (float *)malloc(size);

	float *h_B = (float *)malloc(size);

	float *h_C = (float *)malloc(size);


	if (h_A == NULL || h_B == NULL || h_C == NULL)
	{
		fprintf(stderr, " Ana vektorleri ayirma hatasi!\n");
		exit(EXIT_FAILURE);
	}


	for (int i = 0; i < numElements; ++i)
	{
		h_A[i] = rand() / (float)RAND_MAX;
		h_B[i] = rand() / (float)RAND_MAX;
	}


	float *d_A = NULL;
	err = hipMalloc((void **)&d_A, size);

	if (err != hipSuccess)
	{
		fprintf(stderr, " A device vektorunu ayirma hatasi (hata kodu %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	float *d_B = NULL;
	err = hipMalloc((void **)&d_B, size);

	if (err != hipSuccess)
	{
		fprintf(stderr, "B device vektorunu ayirma hatasi(hata kodu %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	float *d_C = NULL;
	err = hipMalloc((void **)&d_C, size);

	if (err != hipSuccess)
	{
		fprintf(stderr, "C device vektorunu ayirma hatasi (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	printf("Giri� verilerini ana bellekten CUDA cihaz�na kopyalay�n\n");
	err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

	if (err != hipSuccess)
	{
		fprintf(stderr, "A vektoru host tan device a kopyalanamadi(hata kodu %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

	if (err != hipSuccess)
	{
		fprintf(stderr, "B vektoru host tan device a kopyalanamadi (hata kodu %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	int threadsPerBlock = 256;
	int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
	printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
	vectorAdd << <blocksPerGrid, threadsPerBlock >> > (d_A, d_B, d_C, numElements);
	err = hipGetLastError();

	if (err != hipSuccess)
	{
		fprintf(stderr, "VectorAdd �ekirde�i ba�lat�lamad�(hata kodu %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}


	printf("��kt� verilerini CUDA cihaz�ndan ana bilgisayar belle�ine kopyalay�n\n");
	err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

	if (err != hipSuccess)
	{
		fprintf(stderr, "C vektoru device tan host a kopyalanamadi (hata kodu %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}


	for (int i = 0; i < numElements; ++i)
	{
		if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
		{
			fprintf(stderr, "Ogede sonuc dogrulamasi basarisiz oldu %d!\n", i);
			exit(EXIT_FAILURE);
		}
	}

	printf("Test PASSED\n");


	err = hipFree(d_A);

	if (err != hipSuccess)
	{
		fprintf(stderr, "A vektoru bosaltilamadi (hata kodu %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipFree(d_B);

	if (err != hipSuccess)
	{
		fprintf(stderr, "B vektoru bosaltilamadi (hata kodu %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipFree(d_C);

	if (err != hipSuccess)
	{
		fprintf(stderr, "C vektoru bosaltilamadi (hata kodu %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}


	free(h_A);
	free(h_B);
	free(h_C);

	printf("Bitti\n");
	return 0;
}

